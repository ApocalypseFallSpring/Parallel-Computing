#include "hip/hip_runtime.h"
#include "PCFG.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

// CUDA核函数：优化后的口令生成
__global__ void generateGuessesKernel(
    const char* d_guess, 
    int prefix_len,
    const char* d_values,
    const int* d_value_offsets,
    const int* d_value_lengths,
    char* d_output,
    const int* d_output_offsets,
    int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        char* out_ptr = d_output + d_output_offsets[idx];
        
        // 复制密码前缀
        for (int i = 0; i < prefix_len; i++) {
            out_ptr[i] = d_guess[i];
        }
        
        // 复制当前值
        int value_offset = d_value_offsets[idx];
        for (int i = 0; i < d_value_lengths[idx]; i++) {
            out_ptr[prefix_len + i] = d_values[value_offset + i];
        }
    }
}

// 优化后的GPU加速函数
void PriorityQueue::GenerateWithGPU(const std::string& guess, segment* a, int n) {
    // 1. 准备数据 - 优化数据结构减少传输量
    const int prefix_len = guess.length();
    std::vector<int> value_lengths(n);
    std::vector<int> value_offsets(n + 1, 0);
    std::string flat_values;
    
    // 计算值的扁平化存储和偏移量
    for (int i = 0; i < n; ++i) {
        value_lengths[i] = a->ordered_values[i].size();
        value_offsets[i] = flat_values.size();
        flat_values += a->ordered_values[i];
    }
    value_offsets[n] = flat_values.size();  // 结束位置标记
    
    // 计算输出缓冲区大小和偏移
    std::vector<int> output_offsets(n + 1, 0);
    for (int i = 0; i < n; ++i) {
        output_offsets[i + 1] = output_offsets[i] + prefix_len + value_lengths[i];
    }
    const int total_output_size = output_offsets[n];
    
    // 2. 设备内存分配
    char *d_guess = nullptr, *d_values = nullptr, *d_output = nullptr;
    int *d_value_offsets = nullptr, *d_value_lengths = nullptr, *d_output_offsets = nullptr;
    
    hipMalloc(&d_guess, prefix_len);
    hipMalloc(&d_values, flat_values.size());
    hipMalloc(&d_value_offsets, (n + 1) * sizeof(int));
    hipMalloc(&d_value_lengths, n * sizeof(int));
    hipMalloc(&d_output, total_output_size);
    hipMalloc(&d_output_offsets, (n + 1) * sizeof(int));
    
    // 3. 数据传输到GPU
    hipMemcpy(d_guess, guess.c_str(), prefix_len, hipMemcpyHostToDevice);
    hipMemcpy(d_values, flat_values.c_str(), flat_values.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_value_offsets, value_offsets.data(), (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_value_lengths, value_lengths.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_output_offsets, output_offsets.data(), (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    
    // 4. 配置并启动核函数
    const int blockSize = 256;
    const int gridSize = (n + blockSize - 1) / blockSize;
    
    generateGuessesKernel<<<gridSize, blockSize>>>(
        d_guess, prefix_len,
        d_values, d_value_offsets,
        d_value_lengths,
        d_output,
        d_output_offsets,
        n
    );
    
    // 5. 等待核函数执行完成
    hipDeviceSynchronize();
    
    // 6. 将结果复制回主机
    char* h_output = new char[total_output_size];
    hipMemcpy(h_output, d_output, total_output_size, hipMemcpyDeviceToHost);
    
    // 7. 构建最终结果列表
    for (int i = 0; i < n; ++i) {
        guesses.emplace_back(h_output + output_offsets[i], value_lengths[i] + prefix_len);
    }
    
    // 8. 释放资源
    delete[] h_output;
    hipFree(d_guess);
    hipFree(d_values);
    hipFree(d_value_offsets);
    hipFree(d_value_lengths);
    hipFree(d_output);
    hipFree(d_output_offsets);
    
    total_guesses += n;
}

// 修改后的Generate函数
void PriorityQueue::Generate(PT pt) {
    CalProb(pt);

    if (pt.content.size() == 1) {
        segment *a = nullptr;
        if (pt.content[0].type == 1) {
            a = &m.letters[m.FindLetter(pt.content[0])];
        } else if (pt.content[0].type == 2) {
            a = &m.digits[m.FindDigit(pt.content[0])];
        } else if (pt.content[0].type == 3) {
            a = &m.symbols[m.FindSymbol(pt.content[0])];
        }
        
        // 智能选择执行路径
        const int GPU_THRESHOLD = 10000;
        if (a->ordered_values.size() <= GPU_THRESHOLD) {
            for (int i = 0; i < pt.max_indices[0]; i++) {
                guesses.emplace_back(a->ordered_values[i]);
                total_guesses++;
            }
        } else {
            GenerateWithGPU("", a, pt.max_indices[0]);
        }
    } else {
        string guess_prefix;
        int seg_idx = 0;
        for (int idx : pt.curr_indices) {
            if (seg_idx == pt.content.size() - 1) break;
            
            if (pt.content[seg_idx].type == 1) {
                guess_prefix += m.letters[m.FindLetter(pt.content[seg_idx])].ordered_values[idx];
            } else if (pt.content[seg_idx].type == 2) {
                guess_prefix += m.digits[m.FindDigit(pt.content[seg_idx])].ordered_values[idx];
            } else if (pt.content[seg_idx].type == 3) {
                guess_prefix += m.symbols[m.FindSymbol(pt.content[seg_idx])].ordered_values[idx];
            }
            seg_idx++;
        }

        segment *a = nullptr;
        if (pt.content.back().type == 1) {
            a = &m.letters[m.FindLetter(pt.content.back())];
        } else if (pt.content.back().type == 2) {
            a = &m.digits[m.FindDigit(pt.content.back())];
        } else if (pt.content.back().type == 3) {
            a = &m.symbols[m.FindSymbol(pt.content.back())];
        }
        
        // 智能选择执行路径
        const int GPU_THRESHOLD = 10000;
        if (a->ordered_values.size() <= GPU_THRESHOLD) {
            for (int i = 0; i < pt.max_indices.back(); i++) {
                guesses.emplace_back(guess_prefix + a->ordered_values[i]);
                total_guesses++;
            }
        } else {
            GenerateWithGPU(guess_prefix, a, pt.max_indices.back());
        }
    }
}

// 以下为未修改的原始代码部分
void PriorityQueue::CalProb(PT &pt)
{
    // 计算PriorityQueue里面一个PT的流程如下：
    // 1. 首先需要计算一个PT本身的概率。例如，L6S1的概率为0.15
    // 2. 需要注意的是，Queue里面的PT不是“纯粹的”PT，而是除了最后一个segment以外，全部被value实例化的PT
    // 3. 所以，对于L6S1而言，其在Queue里面的实际PT可能是123456S1，其中“123456”为L6的一个具体value。
    // 4. 这个时候就需要计算123456在L6中出现的概率了。假设123456在所有L6 segment中的概率为0.1，那么123456S1的概率就是0.1*0.15

    // 计算一个PT本身的概率。后续所有具体segment value的概率，直接累乘在这个初始概率值上
    pt.prob = pt.preterm_prob;

    // index: 标注当前segment在PT中的位置
    int index = 0;


    for (int idx : pt.curr_indices)
    {
        // pt.content[index].PrintSeg();
        if (pt.content[index].type == 1)
        {
            // 下面这行代码的意义：
            // pt.content[index]：目前需要计算概率的segment
            // m.FindLetter(seg): 找到一个letter segment在模型中的对应下标
            // m.letters[m.FindLetter(seg)]：一个letter segment在模型中对应的所有统计数据
            // m.letters[m.FindLetter(seg)].ordered_values：一个letter segment在模型中，所有value的总数目
            pt.prob *= m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.letters[m.FindLetter(pt.content[index])].total_freq;
            // cout << m.letters[m.FindLetter(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.letters[m.FindLetter(pt.content[index])].total_freq << endl;
        }
        if (pt.content[index].type == 2)
        {
            pt.prob *= m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.digits[m.FindDigit(pt.content[index])].total_freq;
            // cout << m.digits[m.FindDigit(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.digits[m.FindDigit(pt.content[index])].total_freq << endl;
        }
        if (pt.content[index].type == 3)
        {
            pt.prob *= m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx];
            pt.prob /= m.symbols[m.FindSymbol(pt.content[index])].total_freq;
            // cout << m.symbols[m.FindSymbol(pt.content[index])].ordered_freqs[idx] << endl;
            // cout << m.symbols[m.FindSymbol(pt.content[index])].total_freq << endl;
        }
        index += 1;
    }
    // cout << pt.prob << endl;
}

void PriorityQueue::init()
{
    // cout << m.ordered_pts.size() << endl;
    // 用所有可能的PT，按概率降序填满整个优先队列
    for (PT pt : m.ordered_pts)
    {
        for (segment seg : pt.content)
        {
            if (seg.type == 1)
            {
                // 下面这行代码的意义：
                // max_indices用来表示PT中各个segment的可能数目。例如，L6S1中，假设模型统计到了100个L6，那么L6对应的最大下标就是99
                // （但由于后面采用了"<"的比较关系，所以其实max_indices[0]=100）
                // m.FindLetter(seg): 找到一个letter segment在模型中的对应下标
                // m.letters[m.FindLetter(seg)]：一个letter segment在模型中对应的所有统计数据
                // m.letters[m.FindLetter(seg)].ordered_values：一个letter segment在模型中，所有value的总数目
                pt.max_indices.emplace_back(m.letters[m.FindLetter(seg)].ordered_values.size());
            }
            if (seg.type == 2)
            {
                pt.max_indices.emplace_back(m.digits[m.FindDigit(seg)].ordered_values.size());
            }
            if (seg.type == 3)
            {
                pt.max_indices.emplace_back(m.symbols[m.FindSymbol(seg)].ordered_values.size());
            }
        }
        pt.preterm_prob = float(m.preterm_freq[m.FindPT(pt)]) / m.total_preterm;
        // pt.PrintPT();
        // cout << " " << m.preterm_freq[m.FindPT(pt)] << " " << m.total_preterm << " " << pt.preterm_prob << endl;

        // 计算当前pt的概率
        CalProb(pt);
        // 将PT放入优先队列
        priority.emplace_back(pt);
    }
    // cout << "priority size:" << priority.size() << endl;
}

void PriorityQueue::PopNext()
{

    // 对优先队列最前面的PT，首先利用这个PT生成一系列猜测
    Generate(priority.front());

    // 然后需要根据即将出队的PT，生成一系列新的PT
    vector<PT> new_pts = priority.front().NewPTs();
    for (PT pt : new_pts)
    {
        // 计算概率
        CalProb(pt);
        // 接下来的这个循环，作用是根据概率，将新的PT插入到优先队列中
        for (auto iter = priority.begin(); iter != priority.end(); iter++)
        {
            // 对于非队首和队尾的特殊情况
            if (iter != priority.end() - 1 && iter != priority.begin())
            {
                // 判定概率
                if (pt.prob <= iter->prob && pt.prob > (iter + 1)->prob)
                {
                    priority.emplace(iter + 1, pt);
                    break;
                }
            }
            if (iter == priority.end() - 1)
            {
                priority.emplace_back(pt);
                break;
            }
            if (iter == priority.begin() && iter->prob < pt.prob)
            {
                priority.emplace(iter, pt);
                break;
            }
        }
    }

    // 现在队首的PT善后工作已经结束，将其出队（删除）
    priority.erase(priority.begin());
}

vector<PT> PT::NewPTs()
{
    // 存储生成的新PT
    vector<PT> res;

    // 假如这个PT只有一个segment
    // 那么这个segment的所有value在出队前就已经被遍历完毕，并作为猜测输出
    // 因此，所有这个PT可能对应的口令猜测已经遍历完成，无需生成新的PT
    if (content.size() == 1)
    {
        return res;
    }
    else
    {
        // 最初的pivot值。我们将更改位置下标大于等于这个pivot值的segment的值（最后一个segment除外），并且一次只更改一个segment
        // 上面这句话里是不是有没看懂的地方？接着往下看你应该会更明白
        int init_pivot = pivot;

        // 开始遍历所有位置值大于等于init_pivot值的segment
        // 注意i < curr_indices.size() - 1，也就是除去了最后一个segment（这个segment的赋值预留给并行环节）
        for (int i = pivot; i < curr_indices.size() - 1; i += 1)
        {
            // curr_indices: 标记各segment目前的value在模型里对应的下标
            curr_indices[i] += 1;

            // max_indices：标记各segment在模型中一共有多少个value
            if (curr_indices[i] < max_indices[i])
            {
                // 更新pivot值
                pivot = i;
                res.emplace_back(*this);
            }

            // 这个步骤对于你理解pivot的作用、新PT生成的过程而言，至关重要
            curr_indices[i] -= 1;
        }
        pivot = init_pivot;
        return res;
    }

    return res;
}
